#include "hip/hip_runtime.h"
#pragma once

#include "third_party/shared_queue.h"
#include <chrono>
#include <atomic>
#include <thread>
#include <vector>
#include "vec3.h"
#include "triangle.h"
#include "node.h"
#include "debug.cu"
#include "third_party/cuda_helpers/hip/hip_runtime_api.h"

__global__ void computeBoundsAndCentroids(Triangle* triangles, int triangle_count, Vec3* ptr_device_vertex_buffer, int* ptr_device_triangle_ids);

inline int projectToBin(float k_1, float centroid_bin_axis, float scene_min_axis);
inline float cost(int N_L, int N_R, float A_L, float A_R);

//Inputs along the selected axis. E.g. tri_centroid for x,y or z depending on the selected axis.
inline int projectToBin(float k_1, float tri_centroid, float node_min_bounds){
  return min(max((int) (k_1 * (tri_centroid - node_min_bounds)), 0), 16);
}

inline float cost(int N_L, int N_R, float A_L, float A_R){
  return (A_L * N_L + A_R * N_R);
}

__global__ void computeBoundsAndCentroids(Triangle* triangles, int triangle_count, Vec3* ptr_device_vertex_buffer){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= triangle_count)
    return;

  Triangle tri = triangles[node_index];
  Vec3 v0 = ptr_device_vertex_buffer[tri.v0_index];
  Vec3 v1 = ptr_device_vertex_buffer[tri.v1_index];
  Vec3 v2 = ptr_device_vertex_buffer[tri.v2_index];
  Vec3 centroid = (v0 + v1 + v2) / 3.0;
  triangles[node_index].centroid = centroid;

  Vec3 vertex_bounds_min = min(min(v0, v1), v2);
  Vec3 vertex_bounds_max = max(max(v0, v1), v2);
  triangles[node_index].aabb = AABB(vertex_bounds_min, vertex_bounds_max);
}

//See comment in SAHBVH constructor regarding the use of this function.
__global__ void deepCopyTreeToGPU(Node* input_nodes, int nodes_length, Node* output_internal_nodes, Node* output_leaf_nodes, Triangle* triangles, int internal_nodes_count, int leaf_nodes_count, int* triangle_ids){
  //Traverse tree in reverse order and copy nodes to internal_nodes and leaf_nodes buffers.
  int internal_nodes = internal_nodes_count;
  int leaf_nodes = leaf_nodes_count-1;

  for (int i = nodes_length; i >= 0; i--){
    Node node = input_nodes[i];
    if(node.is_leaf){
      // printf("Leaf: Start range: %i, Leaf nodes: %i Nodes left: %i, Internal Nodes %i \n", node.start_range, leaf_nodes, i, internal_nodes);
      node.primitive = &triangles[triangle_ids[node.start_range]];
      output_leaf_nodes[leaf_nodes] = node;
      input_nodes[i].parent = &output_leaf_nodes[leaf_nodes]; //leave a trail for the parent to pick up.
      leaf_nodes--;
    }
    else{
      // printf("Left Child %i, Right Child: %i, Internal Nodes %i, Index: %i\n", node.left_child_i, node.right_child_i, internal_nodes, i);
      node.left_child = input_nodes[node.left_child_i].parent;
      node.right_child = input_nodes[node.right_child_i].parent;
      output_internal_nodes[internal_nodes] = node;
      input_nodes[i].parent = &output_internal_nodes[internal_nodes]; //leave a trail for the parent to pick up.
      internal_nodes--;
    }
  }
}

class SAHBVH{
  Triangle* triangles;
  Triangle* ptr_device_triangles;
  Vec3* ptr_device_vertex_buffer;
  int triangle_count;
  int vertex_count;

  std::atomic_int nodes_created;
  std::atomic_int leaf_nodes_created;

  AABB scene_bounds_triangles;
  AABB scene_bounds_centroids;

  int* triangle_ids;
  int* temp_triangle_ids;
  Node* ptr_host_internal_nodes;

  Node* ptr_device_internal_nodes;
  Node* ptr_device_leaf_nodes;
  Node* ptr_device_temp_nodes;
  int nodes_length;

  SharedQueue<Node*> work_queue;
  std::vector<std::thread> workers;

  public:
  SAHBVH(Triangle* ptr_device_triangles, int triangle_count, Vec3* ptr_device_vertices, int vertex_count, AABB scene_bounds_triangles){
    this->triangle_ids = (int*)malloc(sizeof(int) * triangle_count);
    for (int i = 0; i < triangle_count; i++)
      triangle_ids[i] = i;
    this->temp_triangle_ids = (int*)malloc(sizeof(int) * triangle_count);
    memcpy(temp_triangle_ids, triangle_ids, triangle_count);

    this->nodes_length = 2*triangle_count-1;
    this->ptr_host_internal_nodes = (Node*)calloc(nodes_length, sizeof(Node));
    //Initialize indices to -1, indicating no child is present.
    for (int i = 0; i < nodes_length; i++){
      this->ptr_host_internal_nodes[i].right_child_i = -1;
      this->ptr_host_internal_nodes[i].left_child_i = -1;
    }

    this->nodes_created.store(0);
    this->leaf_nodes_created.store(0);
    
    this->vertex_count = vertex_count;
    this->triangle_count = triangle_count;
    this->scene_bounds_triangles = scene_bounds_triangles;
    this->ptr_device_triangles = ptr_device_triangles;
    this->ptr_device_vertex_buffer = ptr_device_vertices;

    checkCudaErrors(hipMalloc(&ptr_device_internal_nodes, (triangle_count-1) * sizeof(Node)));
    checkCudaErrors(hipMalloc(&ptr_device_leaf_nodes, triangle_count * sizeof(Node)));
    checkCudaErrors(hipMalloc(&ptr_device_temp_nodes, nodes_length * sizeof(Node)));
  }

  ~SAHBVH(){
    free(ptr_host_internal_nodes);
    free(triangle_ids);
    free(temp_triangle_ids);
    checkCudaErrors(hipFree(ptr_device_temp_nodes));
  }

  void splitNode(SAHBVH *bvh, Node* node, Node* nodes, int start, int end, int* triangle_ids, int* temp_triangle_ids, Triangle* triangles, int depth);
  void creationThread(SAHBVH* bvh, int thread_id);

  //Returns device ptr to root of tree.
  Node* construct(){
    using namespace std::chrono;
    // printf("Starting SAH Binning construction.\n");

    computeBoundsAndCentroids<<<triangle_count/64+1, 64>>>(ptr_device_triangles, triangle_count, ptr_device_vertex_buffer);
    checkCudaErrors(hipDeviceSynchronize());
    this->triangles = (Triangle*)malloc(sizeof(Triangle) * triangle_count);
    checkCudaErrors(hipMemcpy(triangles, ptr_device_triangles, sizeof(Triangle) * triangle_count, hipMemcpyDeviceToHost));

    Node* root_node = new Node();
    root_node->start_range = 0;
    root_node->depth = 0;
    root_node->range = triangle_count;
    root_node->aabb = scene_bounds_triangles;
    root_node->left_child_i = 1;
    root_node->right_child_i = 2;
    ptr_host_internal_nodes[0] = *root_node;

    work_queue.push(root_node);
    steady_clock::time_point start = high_resolution_clock::now();
    int num_threads = std::thread::hardware_concurrency();

    for (int i = 0; i < num_threads; i++){
      workers.push_back(std::thread([this, i] {
        this->creationThread(this, i);
      }));
      while(work_queue.size() < num_threads); //HACK: Busy-wait until theres work available for all threads.
    }
    for(int i = 0; i < workers.size(); i++){
      if (workers[i].joinable()){
        workers[i].join();
      }
    }

    steady_clock::time_point stop = high_resolution_clock::now();
    long long duration_ms = duration_cast<milliseconds>(stop - start).count();
    long long duration_us = duration_cast<microseconds>(stop - start).count();
    printf("Binned SAH\t%lli\tms\t%lli\tus\n", duration_ms, duration_us);

    //This memcpy is inevitable if we construct tree on the cpu and render on the gpu.
    checkCudaErrors(hipMemcpy(ptr_device_temp_nodes, ptr_host_internal_nodes, nodes_length * sizeof(Node), hipMemcpyHostToDevice));

    int* ptr_device_triangle_ids;
    checkCudaErrors(hipMalloc(&ptr_device_triangle_ids, sizeof(int) * triangle_count));
    checkCudaErrors(hipMemcpy(ptr_device_triangle_ids, triangle_ids, sizeof(int) * triangle_count, hipMemcpyHostToDevice));
    
    //This could be omitted if the tracing traversal is modified to be compliant with our CPU tree structure.
    //As of right now we're using the same traversal algorithm for both lbvh and binned sah bvh, but the structure is slightly different. 
    //We must therefore first copy the data to the GPU and then update all pointers in the tree to be device, not host.
    deepCopyTreeToGPU<<<1,1>>>(ptr_device_temp_nodes,
      nodes_created.load(),
      ptr_device_internal_nodes,
      ptr_device_leaf_nodes,
      ptr_device_triangles,
      nodes_created.load() - leaf_nodes_created.load(),
      leaf_nodes_created.load(),
      ptr_device_triangle_ids);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(ptr_device_triangle_ids));

    return ptr_device_internal_nodes;
  }
};

void SAHBVH::creationThread(SAHBVH* bvh, int thread_id){
  // int nodes_processed = 0;

  while(!bvh->work_queue.empty()){
    Node* active_node = bvh->work_queue.pop_front();
    splitNode(this,
      active_node,
      this->ptr_host_internal_nodes,
      active_node->start_range,
      active_node->start_range + active_node->range,
      this->triangle_ids,
      this->temp_triangle_ids,
      this->triangles,
      0);
    // nodes_processed++;
  }
  // printf("Thread: %i, Nodes processed: %i\n", thread_id, nodes_processed);
}

void SAHBVH::splitNode(SAHBVH *bvh, Node* node, Node* nodes, int start, int end, int* triangle_ids, int* temp_triangle_ids, Triangle* triangles, int depth){
  const int number_of_bins = 16;
  const int primitive_count = end - start;

  node->aabb           = AABB(Vec3(FLT_MAX, FLT_MAX, FLT_MAX), Vec3(-FLT_MAX, -FLT_MAX, -FLT_MAX));
  AABB centroid_bounds = AABB(Vec3(FLT_MAX, FLT_MAX, FLT_MAX), Vec3(-FLT_MAX, -FLT_MAX, -FLT_MAX));
  
  for(int i = start; i < end; i++){
    node->aabb.join(triangles[triangle_ids[i]].aabb);
    centroid_bounds.join(triangles[triangle_ids[i]].centroid);
  }

  //Decide which axis to sweep. (the longest side).
  int axis;
  Vec3 size = centroid_bounds.max_bounds - centroid_bounds.min_bounds;
  if(size.x() >= size.y() && size.x() >= size.z())
    axis = 0;
  else if(size.y() >= size.x() && size.y() >= size.z())
    axis = 1;
  else
    axis = 2;

  //Compute k_1 for the selected axis.
  float k_1 = number_of_bins * (1.0 - FLT_EPSILON) / 
                (node->aabb.max_bounds.e[axis] - node->aabb.min_bounds.e[axis]);

  //Initialize per bin triangle counts and aabbs.
  int bin_triangle_counts[number_of_bins];
  for (int i = 0; i < number_of_bins; i++)
    bin_triangle_counts[i] = 0;
  AABB bin_aabbs[number_of_bins];
  for (int i = 0; i < number_of_bins; i++)
    bin_aabbs[i] = AABB(Vec3(FLT_MAX, FLT_MAX, FLT_MAX), Vec3(-FLT_MAX, -FLT_MAX, -FLT_MAX));

  //Calculate N_l, N_r, A_l & A_r for all triangles for all bins.
  for(int i = start; i < end; i++){
    int bin_index = projectToBin( k_1, 
                                  triangles[triangle_ids[i]].centroid.e[axis],
                                  centroid_bounds.min_bounds.e[axis]);

    bin_triangle_counts[bin_index]++;
    bin_aabbs[bin_index].join(triangles[triangle_ids[i]].aabb);
  }

  //Sweep from left -->>
  int tri_count_l_sweep[number_of_bins];
  AABB aabb_l_sweep[number_of_bins];

  tri_count_l_sweep[0] = bin_triangle_counts[0];
  aabb_l_sweep[0] = bin_aabbs[0];
  for (int i = 1; i < number_of_bins; i++){
    tri_count_l_sweep[i] = tri_count_l_sweep[i-1] + bin_triangle_counts[i];
    aabb_l_sweep[i] = AABB::join(aabb_l_sweep[i-1], bin_aabbs[i]);
  }

  //Sweep from right <<-- and calculate cost.
  AABB aabb_r_sweep[number_of_bins];
  for (int i = 0; i < number_of_bins - 1; i++)
    aabb_r_sweep[i] =  AABB(Vec3(FLT_MAX, FLT_MAX, FLT_MAX), Vec3(-FLT_MAX, -FLT_MAX, -FLT_MAX));

  aabb_r_sweep[number_of_bins-1] = bin_aabbs[number_of_bins-1];
  float min_cost = FLT_MAX;
  int split_index = 0;

  float bin_width = size.e[axis] / number_of_bins;
  float split_position = centroid_bounds.min_bounds.e[axis] + (bin_width*split_index);

  for (int i = number_of_bins-2; i >= 0; i--){
    int primitives_left = tri_count_l_sweep[i];
    aabb_r_sweep[i] = AABB::join(aabb_r_sweep[i+1], bin_aabbs[i]);
    float sah_cost = cost(  primitives_left,                    //N_L
                            primitive_count - primitives_left,  //N_R
                            aabb_l_sweep[i].surfaceArea(),      //A_L
                            aabb_r_sweep[i].surfaceArea());     //A_R

    if(sah_cost < min_cost){
      min_cost = sah_cost;
      split_index = i;
    }
  }

  split_position = centroid_bounds.min_bounds.e[axis] + (bin_width*split_index);

  //Copy triangle ids to temporary array.
  for (int i = start; i < end; i++)
    temp_triangle_ids[i] = triangle_ids[i];

  //Update triangle id buffers.
  int left_i = start;
  int right_i = end-1;
  for (int i = start; i < end; i++){
    if(triangles[temp_triangle_ids[i]].centroid.e[axis] <= split_position){
      triangle_ids[left_i] = temp_triangle_ids[i];
      left_i++;
    }
    else{
      triangle_ids[right_i] = temp_triangle_ids[i];
      right_i--;
    }
  }
  int split = left_i;

  if(split == start || split == end){
    node->start_range = start;
    node->range = end - start;
    leaf_nodes_created++;
    return;
  }
  node->is_leaf = false;

  //Record node relationships.
  int node_index = ++nodes_created;
  Node* left_child = &nodes[node_index];
  left_child->start_range = start;
  left_child->range = split - start;
  left_child->depth = node->depth + 1;
  left_child->is_leaf = true;
  node->left_child_i = left_child - nodes;
  work_queue.push(left_child);

  node_index = ++nodes_created;
  Node* right_child = &nodes[node_index];
  right_child->start_range = split;
  right_child->range = end - split;
  right_child->is_leaf = true;
  right_child->depth = node->depth + 1;
  node->right_child_i = right_child - nodes;
  work_queue.push(right_child);
}