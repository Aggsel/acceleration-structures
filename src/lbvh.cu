#include "hip/hip_runtime.h"
#pragma once

#include "triangle.h"
#include "node.h"
#include "cuda_helpers/hip/hip_runtime_api.h"

__global__ void constructLBVH(Triangle *triangles, Node* internal_nodes, Node* leaf_nodes, int primitive_count);
__device__ int2 determineRange(Triangle *sorted_morton_codes, int total_primitives, int node_index);
__device__ int findSplit(Triangle *sorted_morton_codes, int first, int last);
__global__ void calculateAABB(Node* internal_nodes, Triangle* leaf_nodes, int leaf_count, Vec3* vert_buff);

__device__ int2 determineRange(Triangle *sorted_morton_codes, int total_primitives, int node_index){
  //Time complexity of the algorithm is proportional to the number of keys covered by the nodes.
  //The widest node is also one that we know in advance:
  if(node_index == 0){
    int2 range;
    range.x = 0;
    range.y = total_primitives-1;
    return range;
  }

  //Determine direction (d).
  //Delta being the number of largest common bits between two keys.
  int current_code = sorted_morton_codes[node_index].morton_code;
  //BUG: Properly handle out of bounds exceptions. (branchless? :O)
  int prev_code = sorted_morton_codes[node_index-1].morton_code;
  int next_code = sorted_morton_codes[node_index+1].morton_code;
  //BUG: Prevent duplicate morton codes.
  // if(prev_code == current_code)
  //   prev_code = prev_code ^ node_index-1;
  // if(next_code == current_code)
  //   next_code = next_code ^ node_index+1;
  int next_delta = __clz(current_code ^ next_code);
  int prev_delta = __clz(current_code ^ prev_code);
  int d = next_delta - prev_delta < 0 ? -1 : 1;

  //Compute upper bound for the length of the range.
  //TODO: __Note from Karras 2012__:
  //      When searching for lmax on
  //      lines 5–8, we have found that it is beneficial to start from a
  //      larger number, e.g. 128, and multiply the value by 4 instead
  //      of 2 after each iteration to reduce the total amount of work.

  int lmax = 2;
  int delta_min = min(next_delta, prev_delta);
  int delta = -1;
  int i = node_index + d * lmax;
  if(i >= 0 && i < total_primitives){
      delta = __clz(current_code ^ sorted_morton_codes[i].morton_code);
  }

  while(delta > delta_min){
      lmax = lmax << 1;
      i = node_index + d * lmax;
      delta = -1;
      if(0 <= i && i < total_primitives)
          delta = __clz(current_code ^ sorted_morton_codes[i].morton_code);
  }

  //Binary search for the other end.
  int l = 0;
  int t = lmax >> 1;
  while(t > 0){
    i = node_index + (l + t) * d;
    delta = -1;
    if(0 <= i && i < total_primitives){
        delta = __clz(current_code ^ sorted_morton_codes[i].morton_code);
    }
    if(delta > delta_min){
        l += t;
    }
    t = t >> 1;
  }
  unsigned int j = node_index + l * d;

  int2 min_max;
  min_max.x = min(node_index, j);
  min_max.y = max(node_index, j);
  return min_max;
}

//From https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
__device__ int findSplit(Triangle *sorted_morton_codes, int first, int last){
  int first_morton = sorted_morton_codes[first].morton_code;
  int last_morton = sorted_morton_codes[last].morton_code;

  if(first_morton == last_morton)
    return (first + last) >> 1;

  //count leading zeros
  int common_prefix = __clz(first_morton ^ last_morton);

  // Use binary search to find where the next bit differs.
  // Specifically, we are looking for the highest object that
  // shares more than common_prefix bits with the first one.
  int split = first;
  int step = last - first;

  do{
    step = (step + 1) >> 1;
    int new_split = split + step;

    if (new_split < last){
      int split_morton = sorted_morton_codes[new_split].morton_code;
      int split_prefix = __clz(first_morton ^ split_morton);
      if (split_prefix > common_prefix)
        split = new_split;
    }
  }while (step > 1);
  
  return split;
}

__global__ void constructLBVH(Triangle *triangles, Node* internal_nodes, Node* leaf_nodes, int primitive_count){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= primitive_count-1)
    return;

  // Find out which range of objects the node corresponds to.
  // (This is where the magic happens!)

  //binary search morton codes.
  int2 range = determineRange(triangles, primitive_count, node_index);
  int first = range.x;
  int last = range.y;

  // Determine where to split the range.
  int split = findSplit(triangles, first, last);

  // Select left_child.
  Node* left_child; // = &internal_nodes[split];
  if(split == first){
    // printf("@BVH::constructLBVH() \tLeaf created during construction.\n"); //@debug
    left_child = &leaf_nodes[split];
    left_child->primitive = &triangles[split];
    left_child->aabb = triangles[split].aabb;
    left_child->isLeaf = true;
  }
  else{
    left_child = &internal_nodes[split];
  }
  left_child->parent = &internal_nodes[node_index];

  // Select right_child.
  Node* right_child; // = &internal_nodes[split + 1];
  if(split + 1 == last){
    // printf("@BVH::constructLBVH() \tLeaf created during construction.\n");  //@debug
    right_child = &leaf_nodes[split + 1];
    right_child->primitive = &triangles[split + 1];
    right_child->aabb = triangles[split + 1].aabb;
    right_child->isLeaf = true;
  }
  else{
    right_child = &internal_nodes[split + 1];
  }
  right_child->parent = &internal_nodes[node_index];

  printf("Node: %i, \tMorton: %i, \tMin: %i, \tMax: %i, \tSplit: %i, \t%i, \t%i\n", node_index, triangles[node_index].morton_code, first, last, split, split == first, split + 1 == last); // @debug

  // Record parent-child relationships.
  internal_nodes[node_index].left_child = left_child;
  internal_nodes[node_index].right_child = right_child;
  // Node 0 is always the root of the tree, the pointer supplied (Node* internalNodes) will be the address of the root.
}

__global__ void calculateAABB(Node* internal_nodes, Node* leaf_nodes, int leaf_count, Vec3* vert_buff, int* counter){
  int leaf_index = blockIdx.x * blockDim.x + threadIdx.x;
  if(leaf_index >= leaf_count)
      return;

  int v0 = leaf_nodes[leaf_index].primitive->v0_index;
  int v1 = leaf_nodes[leaf_index].primitive->v1_index;
  int v2 = leaf_nodes[leaf_index].primitive->v2_index;

  Vec3 min_bounds = min(min(vert_buff[v0], vert_buff[v1]), vert_buff[v2]);
  Vec3 max_bounds = max(max(vert_buff[v0], vert_buff[v1]), vert_buff[v2]);
  AABB leaf_aabb;
  leaf_aabb.min_bounds = min_bounds;
  leaf_aabb.max_bounds = max_bounds;

  Node* current_node_ptr = leaf_nodes[leaf_index].parent;

  if(!current_node_ptr){
    printf("\nTHIS SHOULD NEVER HAPPEN OH GOD WHY\n");
    return;
  }

  current_node_ptr->aabb = leaf_aabb; //BUG: <- sometimes hipErrorIllegalAddress

  while(true){
    if(!current_node_ptr){
      AABB aabb = internal_nodes[0].aabb;
      printf("@BVH::constructAABB() \t internal_nodes[0]: 0x%p internal_nodes: 0x%p Min Bounds: (%f, %f, %f)\n", internal_nodes, internal_nodes, aabb.min_bounds.x(), aabb.min_bounds.y(), aabb.min_bounds.z());  //@debug
      printf("@BVH::constructAABB() \t internal_nodes[0]: 0x%p internal_nodes: 0x%p Max Bounds: (%f, %f, %f)\n", internal_nodes, internal_nodes, aabb.max_bounds.x(), aabb.max_bounds.y(), aabb.max_bounds.z());  //@debug
      return; //Root reached, return.
    }

    int parent_index = current_node_ptr - internal_nodes;
    int old = atomicCAS(&counter[parent_index], 0, 1);
    if(old == 0){ //This thread reached the node first. 
      return;
    }

    // if(current_node_ptr->left_child != nullptr)
    //   current_node_ptr->aabb.join(current_node_ptr->left_child->aabb);
    // if(current_node_ptr->right_child != nullptr)
    //   current_node_ptr->aabb.join(current_node_ptr->right_child->aabb);
    current_node_ptr->aabb = AABB::join(current_node_ptr->left_child->aabb,
                                        current_node_ptr->right_child->aabb);

    // printf("Joined min(%f, %f, %f) max(%f, %f, %f) \t&& min(%f, %f, %f) max(%f, %f, %f) \t = min(%f, %f, %f) max(%f, %f, %f)\n",
    // current_node_ptr->left_child->aabb.min_bounds.x(),
    // current_node_ptr->left_child->aabb.min_bounds.y(),
    // current_node_ptr->left_child->aabb.min_bounds.z(),
    // current_node_ptr->left_child->aabb.max_bounds.x(),
    // current_node_ptr->left_child->aabb.max_bounds.y(),
    // current_node_ptr->left_child->aabb.max_bounds.z(),
    // current_node_ptr->right_child->aabb.min_bounds.x(),
    // current_node_ptr->right_child->aabb.min_bounds.y(),
    // current_node_ptr->right_child->aabb.min_bounds.z(),
    // current_node_ptr->right_child->aabb.max_bounds.x(),
    // current_node_ptr->right_child->aabb.max_bounds.y(),
    // current_node_ptr->right_child->aabb.max_bounds.z(),
    // current_node_ptr->aabb.min_bounds.x(),
    // current_node_ptr->aabb.min_bounds.y(),
    // current_node_ptr->aabb.min_bounds.z(),
    // current_node_ptr->aabb.max_bounds.x(),
    // current_node_ptr->aabb.max_bounds.y(),
    // current_node_ptr->aabb.max_bounds.z() );

    if(!current_node_ptr->parent)   //Parent does not exist, we should be at the root.
      printf("\nCurrent node (root): %p\n", current_node_ptr);

    current_node_ptr = current_node_ptr->parent;
  }
  
  //  Relevant resources:
  //  https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
  //  https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
}

//https://developer.nvidia.com/blog/thinking-parallel-part-ii-tree-traversal-gpu/
//TODO: This is not optimized AT ALL.
__global__ void traverseTree(Node* root){
  printf("\nTraversing tree...\n");
  Node* stack[64];
  Node** stackPtr = stack;

  stackPtr++;
  *stackPtr = (Node*)NULL;

  int i = 0;
  int leaf_count = 0;
  Node* node = root;
  do{
    Node* left_child = node->left_child;
    Node* right_child = node->left_child;

    //Only continue traversing children if they're no leaves.
    //TODO: This is where we can check for ray aabb intersections. 
    bool traverseL = !left_child->isLeaf;
    bool traverseR = !right_child->isLeaf;

    if (!traverseL && !traverseR){
      leaf_count++;
      node = *--stackPtr;
    }
    else{
      node = (traverseL) ? left_child : right_child;
      if (traverseL && traverseR)
        *stackPtr++ = right_child;
    }
    i++;
  }while (node != nullptr);

  printf("Traversed %i nodes, %i leaves found.\n", i, leaf_count);
}

class BVH{
  Node* ptr_device_internal_nodes;
  Node* ptr_device_leaf_nodes;

  Triangle* ptr_device_triangles;
  int triangle_count;
  Vec3* ptr_device_vertices;
  int vertex_count;

  int* ptr_device_visited_node_counters;

  public:
  //BUG: We have no guarantee that the triangle or vertex pointer can be dereferenced 
  //     safely for the entire lifetime of this object.
  BVH(Triangle* ptr_device_triangles, int triangle_count, Vec3* ptr_device_vertices, int vertex_count){
    this->ptr_device_triangles = ptr_device_triangles;
    this->triangle_count = triangle_count;

    this->ptr_device_vertices = ptr_device_vertices;
    this->vertex_count = vertex_count;

    checkCudaErrors(hipMalloc(&ptr_device_internal_nodes, (triangle_count-1)*sizeof(Node)));
    checkCudaErrors(hipMalloc(&ptr_device_leaf_nodes, (triangle_count)*sizeof(Node)));

    checkCudaErrors(hipMalloc(&ptr_device_visited_node_counters, (triangle_count-1)));
    //There's no calloc equivalent for cuda. https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
    checkCudaErrors(hipMemset(ptr_device_visited_node_counters, 0, (triangle_count-1)));
  }

  ~BVH(){
    //BUG: We might want to leave the responsibility of deallocation to the caller of BVH::construct().
    //     This could result in a nullpt when trying to access the tree.
    checkCudaErrors(hipFree(ptr_device_internal_nodes));
    checkCudaErrors(hipFree(ptr_device_leaf_nodes));
    checkCudaErrors(hipFree(ptr_device_visited_node_counters));
  }

  //Returns device ptr to root of tree.
  Node* construct(){
    //TODO: Move morton code generation, scene bounding box calculation etc to this function.
    // <<<x,y>>> Launches x thread blocks with y threads per block.
    constructLBVH<<<(triangle_count-1)/64+1,64>>>(ptr_device_triangles, ptr_device_internal_nodes, ptr_device_leaf_nodes, triangle_count);
    checkCudaErrors(hipDeviceSynchronize());

    calculateAABB<<<triangle_count/64+1,64>>>(ptr_device_internal_nodes, ptr_device_leaf_nodes, triangle_count, ptr_device_vertices, ptr_device_visited_node_counters);
    checkCudaErrors(hipDeviceSynchronize());

    traverseTree<<<1,1>>>(ptr_device_internal_nodes);
    checkCudaErrors(hipDeviceSynchronize());

    return ptr_device_internal_nodes;
  }
};