#include "hip/hip_runtime.h"
#pragma once

#include <thrust/sort.h>
#include "triangle.h"
#include "node.h"
#include "third_party/cuda_helpers/hip/hip_runtime_api.h"

__global__ void constructLBVH(Triangle *triangles, Node* internal_nodes, Node* leaf_nodes, int primitive_count);
__global__ void calculateAABB(Node* internal_nodes, Triangle* leaf_nodes, int leaf_count, Vec3* vert_buff);
__global__ void generateMortonCodes(Triangle* triangles, int triangle_count, Vec3* ptr_device_vertex_buffer, Vec3 scene_bounds_min, Vec3 inverse_min_max);
__device__ int2 determineRange(Triangle *sorted_morton_codes, int total_primitives, int node_index);
__device__ int findSplit(Triangle *sorted_morton_codes, int first, int last);
__device__ unsigned int mortonCode(Vec3 v);
__device__ inline unsigned int expandBits(unsigned int v);

// https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
__device__ __host__ inline unsigned int expandBits(unsigned int v){
  v = (v * 0x00010001u) & 0xFF0000FFu;
  v = (v * 0x00000101u) & 0x0F00F00Fu;
  v = (v * 0x00000011u) & 0xC30C30C3u;
  v = (v * 0x00000005u) & 0x49249249u;
  return v;
}

// https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
//Expects an input Vec3(0..1, 0..1, 0..1)
__device__ unsigned int mortonCode(Vec3 v){
  //Clamp coordinates to 10 bits.
  float x = min(max(v.x() * 1024.0f, 0.0f), 1023.0f);
  float y = min(max(v.y() * 1024.0f, 0.0f), 1023.0f);
  float z = min(max(v.z() * 1024.0f, 0.0f), 1023.0f);
  //Bit shift componentwise before merging bits into morton code.
  unsigned int xx = expandBits((unsigned int)x) << 2;
  unsigned int yy = expandBits((unsigned int)y) << 1;
  unsigned int zz = expandBits((unsigned int)z);
  return xx | yy | zz;
}

__global__ void generateMortonCodes(Triangle* triangles, int triangle_count, Vec3* ptr_device_vertex_buffer, Vec3 scene_bounds_min, Vec3 inverse_min_max){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= triangle_count-1)
    return;

  Triangle tri = triangles[node_index];
  Vec3 v0 = ptr_device_vertex_buffer[tri.v0_index];
  Vec3 v1 = ptr_device_vertex_buffer[tri.v1_index];
  Vec3 v2 = ptr_device_vertex_buffer[tri.v2_index];
  Vec3 centroid = (v0 + v1 + v2) / 3.0;

  centroid.e[0] = (centroid.x() - scene_bounds_min.x()) * inverse_min_max.x();
  centroid.e[1] = (centroid.y() - scene_bounds_min.y()) * inverse_min_max.y();
  centroid.e[2] = (centroid.z() - scene_bounds_min.z()) * inverse_min_max.z();
  triangles[node_index].morton_code = mortonCode(centroid);

  Vec3 vertex_bounds_min = min(min(v0, v1), v2);
  Vec3 vertex_bounds_max = max(max(v0, v1), v2);
  triangles[node_index].aabb = AABB(vertex_bounds_min, vertex_bounds_max);
}

__device__ int commonPrefix(Triangle *morton_codes, int index1, int index2){
  unsigned int key1 = morton_codes[index1].morton_code;
  unsigned int key2 = morton_codes[index2].morton_code;
  if(key1 != key2)
    return __clz(key1 ^ key2);
  return __clz(index1 ^ index2);
}

__device__ int2 determineRange(Triangle *sorted_morton_codes, int total_primitives, int node_index){
  //Time complexity of the algorithm is proportional to the number of keys covered by the nodes.
  //The widest node is also one that we know in advance:
  if(node_index == 0){
    int2 range;
    range.x = 0;
    range.y = total_primitives-1;
    return range;
  }

  //Determine direction (d).
  //Delta being the number of largest common bits between two keys.
  int next_delta = commonPrefix(sorted_morton_codes, node_index, node_index+1);
  int prev_delta = commonPrefix(sorted_morton_codes, node_index, node_index-1);
  int d = next_delta - prev_delta < 0 ? -1 : 1;

  //Compute upper bound for the length of the range.
  int lmax = 128;
  int delta_min = min(next_delta, prev_delta);
  int delta = -1;
  int i = node_index + d * lmax;
  if(i >= 0 && i < total_primitives){
    delta = commonPrefix(sorted_morton_codes, node_index, i);
  }

  while(delta > delta_min){
    lmax = lmax << 2;
    i = node_index + d * lmax;
    delta = -1;
    if(0 <= i && i < total_primitives)
      delta = commonPrefix(sorted_morton_codes, node_index, i);
  }

  //Binary search for the other end.
  int l = 0;
  int t = lmax >> 1;
  while(t > 0){
    i = node_index + (l + t) * d;
    delta = -1;

    if(0 <= i && i < total_primitives)
      delta = commonPrefix(sorted_morton_codes, node_index, i);

    if(delta > delta_min)
      l += t;

    t = t >> 1;
  }
  unsigned int j = node_index + l * d;

  int2 min_max;
  min_max.x = min(node_index, j);
  min_max.y = max(node_index, j);
  return min_max;
}

//From https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
__device__ int findSplit(Triangle *sorted_morton_codes, int first, int last){
  int first_morton = sorted_morton_codes[first].morton_code;
  int last_morton = sorted_morton_codes[last].morton_code;

  if(first_morton == last_morton)
    return (first + last) >> 1;

  //count leading zeros
  int common_prefix = commonPrefix(sorted_morton_codes, first, last);

  // Use binary search to find where the next bit differs.
  // Specifically, we are looking for the highest object that
  // shares more than common_prefix bits with the first one.
  int split = first;
  int step = last - first;

  do{
    step = (step + 1) >> 1;
    int new_split = split + step;

    if (new_split < last){
      int split_prefix = commonPrefix(sorted_morton_codes, first, new_split);
      if (split_prefix > common_prefix)
        split = new_split;
    }
  }while (step > 1);
  
  return split;
}

__global__ void constructLBVH(Triangle *triangles, Node* internal_nodes, Node* leaf_nodes, int primitive_count){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= primitive_count-1)
    return;

  //binary search morton codes.
  int2 range = determineRange(triangles, primitive_count, node_index);
  int first = range.x;
  int last = range.y;

  // Determine where to split the range.
  int split = findSplit(triangles, first, last);

  // Select left_child.
  Node* left_child;
  if(split == first){
    left_child = &leaf_nodes[split];
    left_child->primitive = &triangles[split];
    left_child->aabb = triangles[split].aabb;
    left_child->isLeaf = true;
  }
  else{
    left_child = &internal_nodes[split];
    left_child->isLeaf = false;
  }
  
  // Select right_child.
  Node* right_child;
  if(split + 1 == last){
    right_child = &leaf_nodes[split + 1];
    right_child->primitive = &triangles[split + 1];
    right_child->aabb = triangles[split + 1].aabb;
    right_child->isLeaf = true;
  }
  else{
    right_child = &internal_nodes[split + 1];
    right_child->isLeaf = false;
  }

  // printf("Node: %i, \tMorton: %i, \tMin: %i, \tMax: %i, \tSplit: %i, \t%i, \t%i\n", node_index, triangles[node_index].morton_code, first, last, split, split == first, split + 1 == last); // @debug
  Node *self_ptr = &internal_nodes[node_index];

  assert(self_ptr != nullptr);

  right_child->parent = self_ptr;
  left_child->parent = self_ptr;
  self_ptr->left_child = left_child;
  self_ptr->right_child = right_child;

  // Node 0 is always the root of the tree, the pointer supplied (Node* internalNodes) will be the address of the root.
}

//In parallel, traverse the tree from each leaf upwards.
//The first execution thread to reach a node returns. 
//The second thread joins the nodes AABB and continues traversal.
__global__ void calculateAABB(Node* internal_nodes, Node* leaf_nodes, int leaf_count, Vec3* vert_buff, int* counter){
  int leaf_index = blockIdx.x * blockDim.x + threadIdx.x;
  if(leaf_index >= leaf_count)
      return;

  //Calculate leaf AABB.
  Triangle *leaf_primitive = leaf_nodes[leaf_index].primitive;
  int v0 = leaf_primitive->v0_index;
  int v1 = leaf_primitive->v1_index;
  int v2 = leaf_primitive->v2_index;

  Vec3 min_bounds = min(min(vert_buff[v0], vert_buff[v1]), vert_buff[v2]);
  Vec3 max_bounds = max(max(vert_buff[v0], vert_buff[v1]), vert_buff[v2]);
  AABB leaf_aabb;
  leaf_aabb.min_bounds = min_bounds;
  leaf_aabb.max_bounds = max_bounds;

  Node* parent_node = leaf_nodes[leaf_index].parent;
  assert(parent_node != nullptr);

  parent_node->aabb = leaf_aabb; //BUG: <- sometimes hipErrorIllegalAddress, due to duplicate morton codes.

  while(true){
    if(parent_node == nullptr)  //Root reached.
      return;

    int parent_index = parent_node - internal_nodes;
    int old = atomicCAS(&counter[parent_index], 0, 1);
    if(old == 0){ //This thread reached the node first. 
      return;
    }

    parent_node->aabb = AABB::join(parent_node->left_child->aabb,
                                        parent_node->right_child->aabb);

    parent_node = parent_node->parent;
  }
  
  //  Relevant resources:
  //  https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
  //  https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
}

// Modified from 
// https://developer.nvidia.com/blog/thinking-parallel-part-ii-tree-traversal-gpu/
// TODO: This is not optimized AT ALL.
__global__ void traverseTree(Node* root){
  printf("\nTraversing tree...\n");
  Node* stack[128];
  int stack_index = -1;
  stack_index++;
  stack[stack_index] = nullptr;

  int i = 0;
  Node* node = root;
  do{
    Node* left_child = node->left_child;
    Node* right_child = node->right_child;

    //Only continue traversing children if they're no leaves.
    //TODO: This is where we can check for ray aabb intersections. 
    bool traverse_left  = !left_child ->isLeaf;
    bool traverse_right = !right_child->isLeaf;

    if(!traverse_left && !traverse_right){
      node = stack[stack_index];
      stack_index--;
    }
    else{
      //Prioritize traversing left branch.
      node = traverse_left ? left_child : right_child;

      //Push right child onto the stack if both branches should be traversed.
      if (traverse_left && traverse_right){
        stack_index++;
        stack[stack_index] = right_child;
      }
    }
    i++;
  }while(node != nullptr);
}

__global__ void printInternalNodes(Node* internal_nodes, int primitive_count, Triangle* leaf_nodes){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= primitive_count-1)
    return;
  Node* node = &internal_nodes[node_index];
  printf("Node: %i\tNode: 0x%p\tParent: 0x%p, \tLeft Child: 0x%p, \tRight Child: 0x%p\tMorton: %i\n", node_index, node, node->parent, node->left_child, node->right_child, leaf_nodes[node_index].morton_code);
}

__global__ void printLeafNodes(Node* leaf_nodes, int primitive_count, Triangle* triangles){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= primitive_count)
    return;
  Node* node = &leaf_nodes[node_index];
  printf("Leaf: \tNode: %i\tNode: 0x%p\tParent: 0x%p, \tLeft Child: 0x%p, \tRight Child: 0x%p\tMorton: %i\n", node_index, node, node->parent, node->left_child, node->right_child, triangles[node_index].morton_code);
}

class LBVH{
  Node* ptr_device_internal_nodes;
  Node* ptr_device_leaf_nodes;

  Triangle* ptr_device_triangles;
  int triangle_count;
  Vec3* ptr_device_vertices;
  int vertex_count;

  int* ptr_device_visited_node_counters;

  AABB scene_bounds;

  void populateMortonCodes(){
    int threads_per_block = 512;
    Vec3 inverse_min_max = 1.0/(scene_bounds.max_bounds - scene_bounds.min_bounds);
    generateMortonCodes<<<triangle_count/threads_per_block+1, threads_per_block>>>(ptr_device_triangles, triangle_count, ptr_device_vertices, scene_bounds.min_bounds, inverse_min_max);
    checkCudaErrors(hipDeviceSynchronize());
  }

  public:
  LBVH(Triangle* ptr_device_triangles, int triangle_count, Vec3* ptr_device_vertices, int vertex_count, AABB scene_bounds){
    this->ptr_device_triangles = ptr_device_triangles;
    this->triangle_count = triangle_count;

    this->ptr_device_vertices = ptr_device_vertices;
    this->vertex_count = vertex_count;

    this->scene_bounds = scene_bounds;

    // Allocate and initialize memory for:
    // BVH internal nodes, leaf nodes and our AABB bottom up traversal counter buffer.

    checkCudaErrors(hipMalloc(&ptr_device_internal_nodes, (triangle_count-1)*sizeof(Node)));
    checkCudaErrors(hipMalloc(&ptr_device_leaf_nodes, (triangle_count)*sizeof(Node)));
    //There's no calloc equivalent for cuda. https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html
    checkCudaErrors(hipMemset(ptr_device_internal_nodes, 0, (triangle_count-1)*sizeof(Node)));
    checkCudaErrors(hipMemset(ptr_device_leaf_nodes, 0, (triangle_count)*sizeof(Node)));

    checkCudaErrors(hipMalloc(&ptr_device_visited_node_counters, (triangle_count-1)));
    checkCudaErrors(hipMemset(ptr_device_visited_node_counters, 0, (triangle_count-1)));
  }

  ~LBVH(){
    checkCudaErrors(hipFree(ptr_device_internal_nodes));
    checkCudaErrors(hipFree(ptr_device_leaf_nodes));
    checkCudaErrors(hipFree(ptr_device_visited_node_counters));
  }

  //Returns device ptr to root of tree.
  Node* construct(){
    //TODO: Move morton code generation, scene bounding box calculation etc to this function.
    // <<<x,y>>> Launches x thread blocks with y threads per block.
    const int threads_per_block = 512;
    populateMortonCodes();

    // ----------- SORT -----------
    // Sorts the triangle buffer based on the computed morton codes. (using < overloading from the triangle struct).
    thrust::sort(thrust::device, ptr_device_triangles, ptr_device_triangles+triangle_count);

    constructLBVH<<<(triangle_count-1)/threads_per_block+1, threads_per_block>>>(ptr_device_triangles, ptr_device_internal_nodes, ptr_device_leaf_nodes, triangle_count);
    checkCudaErrors(hipDeviceSynchronize());

    calculateAABB<<<triangle_count/threads_per_block+1, threads_per_block>>>(ptr_device_internal_nodes, ptr_device_leaf_nodes, triangle_count, ptr_device_vertices, ptr_device_visited_node_counters);
    checkCudaErrors(hipDeviceSynchronize());

    // --- @Debug purposes ---
    // printInternalNodes<<<(triangle_count-1)/threads_per_block+1, threads_per_block>>>(ptr_device_internal_nodes, triangle_count, ptr_device_triangles);
    // checkCudaErrors(hipDeviceSynchronize());
    // printLeafNodes<<<triangle_count/threads_per_block+1, threads_per_block>>>(ptr_device_leaf_nodes, triangle_count, ptr_device_triangles);
    // checkCudaErrors(hipDeviceSynchronize());
    // traverseTree<<<1,1>>>(ptr_device_internal_nodes);
    // checkCudaErrors(hipDeviceSynchronize());

    printf("LBVH Construction completed.\n");
    return ptr_device_internal_nodes;
  }
};