#include "hip/hip_runtime.h"
#define TINYOBJLOADER_IMPLEMENTATION 
#define PI 3.1415926535897932385
#define EPSILON 0.000001

#include <iostream>

#include "aabb.h"
#include "node.h"
#include "triangle.h"
#include "vec3.h"
#include "vec2.h"
#include "raytracer/render_config.h"
#include "raytracer/ray.h"
#include "raytracer/hit.h"
#include "raytracer/camera.h"
#include "raytracer/raytracer.cuh"
#include "image.h"
#include "main.h"
#include "math_util.h"

#include "lbvh.cu"
#include "sahbvh.cu"

#include "third_party/cuda_helpers/hip/hip_runtime_api.h"      //checkCudaErrors
#include "third_party/tiny_obj_loader.h"

enum BVH_Type{ none, lbvh, sahbvh };

int main(int argc, char *argv[]){
  std::string filename = "sample_models/large_210.obj";
  int samples_per_pixel = 30;
  int image_height = 512;
  int image_width = 512;
  int max_bounces = 5;
  char* output_filename = "output.ppm";
  BVH_Type bvh_type = lbvh;

  // ----------- CL ARGUMENTS  -----------
  for (size_t i = 2; i < argc; i+=2){
    char* flag = argv[i-1];
    char* parameter = argv[i];
    if(!strcmp(flag, "-i") ||   !strcmp(flag, "--input"))
      filename = std::string(parameter);
    if(!strcmp(flag, "-o") ||   !strcmp(flag, "--image-output"))
      output_filename = parameter;
    if(!strcmp(flag, "-spp") || !strcmp(flag, "--samples-per-pixel"))
      samples_per_pixel = atoi(parameter);
    if(!strcmp(flag, "-iw") ||  !strcmp(flag, "--image-width"))
      image_width = atoi(parameter);
    if(!strcmp(flag, "-ih") ||  !strcmp(flag, "--image-height"))
      image_height = atoi(parameter);
    if(!strcmp(flag, "--max-depth"))
      max_bounces = atoi(parameter);
    if(!strcmp(flag, "-bvh")){
      bvh_type = (BVH_Type)atoi(parameter);
    }
  }

  // ----------- LOAD SCENE  -----------
  tinyobj::ObjReaderConfig reader_config;
  tinyobj::ObjReader reader;

  if (!reader.ParseFromFile(filename, reader_config)) {
    if (!reader.Error().empty()) {
        std::cerr << "TinyObjReader: " << reader.Error();
    }
    exit(1);
  }

  if (!reader.Warning().empty()) {
    std::cout << "TinyObjReader: " << reader.Warning();
  }

  const tinyobj::attrib_t &attrib = reader.GetAttrib();
  const std::vector<tinyobj::shape_t> &shapes = reader.GetShapes();

  std::cout << "\nFile '" << filename << "' loaded." << std::endl;
  int vertex_count = (int)(attrib.vertices.size()) / 3;
  printf("\t# vertices        = %d\n", vertex_count);
  int indices_count = (int)(shapes[0].mesh.indices.size());
  printf("\t# vertex indices  = %d\n", indices_count);
  int normals_count = (int)(attrib.normals.size()) / 3;
  printf("\t# normals         = %d\n", normals_count);
  int poly_count = indices_count / 3;
  printf("\t# triangles       = %d\n\n", poly_count);

  // ------------ Scene bounding box -----------------
  Vec3 min_bounds = Vec3( 100000000.0, 100000000.0,   100000000.0);
  Vec3 max_bounds = Vec3(-100000000.0,-100000000.0,  -100000000.0);

  for (int i = 0; i < attrib.vertices.size(); i+=3){
    float x = attrib.vertices[i  ];
    float y = attrib.vertices[i+1];
    float z = attrib.vertices[i+2];
    min_bounds.e[0] = min(min_bounds.x(), x);
    min_bounds.e[1] = min(min_bounds.y(), y);
    min_bounds.e[2] = min(min_bounds.z(), z);

    max_bounds.e[0] = max(max_bounds.x(), x);
    max_bounds.e[1] = max(max_bounds.y(), y);
    max_bounds.e[2] = max(max_bounds.z(), z);
  }
  AABB scene_bounding_box(min_bounds, max_bounds);
  printf("Scene bounds calculated...\n\tMin Bounds: (%f, %f, %f)\n", min_bounds.x(), min_bounds.y(), min_bounds.z());
  printf("\tMax Bounds: (%f, %f, %f)\n", max_bounds.x(), max_bounds.y(), max_bounds.z());  


  //The Obj reader does not store vertex indices in contiguous memory.
  //Copy the indices into a block of memory on the host device.
  //This is required beforehand regardless och BVH construction method.
  Triangle *ptr_host_triangles = (Triangle*)malloc(sizeof(Triangle) * poly_count);
  for (int i = 0; i < indices_count; i+=3){
    Triangle tempTri = Triangle();
    int v0_index = shapes[0].mesh.indices[i  ].vertex_index;
    int v1_index = shapes[0].mesh.indices[i+1].vertex_index;
    int v2_index = shapes[0].mesh.indices[i+2].vertex_index;
    tempTri.v0_index = v0_index;
    tempTri.v1_index = v1_index;
    tempTri.v2_index = v2_index;
    ptr_host_triangles[i/3] = tempTri;
  }

  //Allocate and memcpy index, vertex and normal buffers from host to device.
  Triangle *ptr_device_triangles = nullptr;
  hipMalloc(&ptr_device_triangles, poly_count * sizeof(Triangle));
  hipMemcpy(ptr_device_triangles, ptr_host_triangles, poly_count * sizeof(Triangle), hipMemcpyHostToDevice);

  Vec3 *ptr_device_vertices = nullptr;
  hipMalloc(&ptr_device_vertices, vertex_count * sizeof(Vec3));
  hipMemcpy(ptr_device_vertices, attrib.vertices.data(), vertex_count * sizeof(Vec3), hipMemcpyHostToDevice);

  Vec3 *ptr_device_normals = nullptr;
  hipMalloc(&ptr_device_normals, normals_count * sizeof(Vec3));
  hipMemcpy(ptr_device_normals, attrib.normals.data(), normals_count * sizeof(Vec3), hipMemcpyHostToDevice);


  // ----------- CONSTRUCT Karras 2012 -----------
  LBVH lbvh(ptr_device_triangles, poly_count, ptr_device_vertices, vertex_count, scene_bounding_box);
  Node* ptr_device_tree = lbvh.construct();

  // ----------- RENDER -----------
  RenderConfig config(image_width, image_height, samples_per_pixel, max_bounces, 1337);
  Camera cam = Camera(config.img_width, config.img_height, 90.0f, 1.0f, Vec3(0,0,0));
  Raytracer raytracer = Raytracer(config, ptr_device_vertices, ptr_device_normals, ptr_device_triangles, indices_count);
  printf("Starting rendering...\n");
  Vec3* ptr_device_img = nullptr;
  switch(bvh_type){
    case BVH_Type::none:
      ptr_device_img = raytracer.render(cam);
      break;
    case BVH_Type::lbvh || BVH_Type::sahbvh:
      ptr_device_img = raytracer.renderBVH(ptr_device_tree, cam);
      break;
  }
  printf("Render complete.\n");

  //Copy framebuffer to host and save to disk.
  Image render_output = Image(config.img_width, config.img_height);
  render_output.copyFromDevice(ptr_device_img, config.img_height * config.img_width);
  render_output.save(output_filename);

  free(ptr_host_triangles);
  checkCudaErrors(hipFree(ptr_device_triangles));
  checkCudaErrors(hipFree(ptr_device_vertices));
  checkCudaErrors(hipFree(ptr_device_normals));
  return 0;
}