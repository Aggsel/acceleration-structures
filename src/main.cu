#include "hip/hip_runtime.h"
#define TINYOBJLOADER_IMPLEMENTATION 
#define PI 3.1415926535897932385
#define EPSILON FLT_EPSILON

#include <iostream>
#include <chrono>

#include "aabb.h"
#include "node.h"
#include "triangle.h"
#include "vec3.h"
#include "vec2.h"
#include "raytracer/render_config.h"
#include "raytracer/ray.h"
#include "raytracer/hit.h"
#include "raytracer/camera.h"
#include "raytracer/raytracer.cuh"
#include "image.h"
#include "obj_loader.h"
#include "math_util.h"

#include "lbvh.cu"
#include "sahbvh.cu"

enum class BVH_Type{NONE = 0, LBVH = 1, SAHBVH = 2};
enum class Render_Type{NONE = 0, NORMAL = 1, HEATMAP = 2};

int main(int argc, char *argv[]){
  std::string filename = "sample_models/large_11k.obj";
  int samples_per_pixel = 30;
  int image_height = 512;
  int image_width = 512;
  int max_bounces = 5;
  float cam_x = 0.0;
  float cam_y = 0.0;
  float cam_z = 0.0;
  Render_Type render_type = Render_Type::NORMAL;
  char* output_filename = "output.ppm";
  BVH_Type bvh_type = BVH_Type::LBVH;

  // ----------- CL ARGUMENTS  -----------
  for (size_t i = 2; i < argc; i+=2){
    char* flag = argv[i-1];
    char* parameter = argv[i];
    if(!strcmp(flag, "-i") ||   !strcmp(flag, "--input"))
      filename = std::string(parameter);
    if(!strcmp(flag, "-o") ||   !strcmp(flag, "--image-output"))
      output_filename = parameter;
    if(!strcmp(flag, "-spp") || !strcmp(flag, "--samples-per-pixel"))
      samples_per_pixel = atoi(parameter);
    if(!strcmp(flag, "-iw") ||  !strcmp(flag, "--image-width"))
      image_width = atoi(parameter);
    if(!strcmp(flag, "-ih") ||  !strcmp(flag, "--image-height"))
      image_height = atoi(parameter);
    if(!strcmp(flag, "--max-depth"))
      max_bounces = atoi(parameter);
    if(!strcmp(flag, "-bvh"))
      bvh_type = (BVH_Type)atoi(parameter);
    if(!strcmp(flag, "-r") || !strcmp(flag, "--render"))
      render_type = (Render_Type)atoi(parameter);
    if(!strcmp(flag, "-x"))
      cam_x = atof(parameter);
    if(!strcmp(flag, "-y"))
      cam_y = atof(parameter);
    if(!strcmp(flag, "-z"))
      cam_z = atof(parameter);
  }

  //Try to read .obj from disk and create necessary geometry buffers on the GPU.
  ObjLoader obj(filename);
  AABB scene_bounding_box        = obj.getSceneBoundingBox();
  Triangle* ptr_device_triangles = obj.createDeviceTriangleBuffer();
  Vec3* ptr_device_vertices      = obj.createDeviceVertexBuffer();
  Vec3* ptr_device_normals       = obj.createDeviceNormalBuffer();

  //BUG/Minor Issue: Calling the constructor for both of these classes will allocate 
  //                 device memory that might not be used.
  LBVH lbvh(ptr_device_triangles, obj.triangle_count, ptr_device_vertices, obj.vertex_count, scene_bounding_box);
  SAHBVH sahbvh(ptr_device_triangles, obj.triangle_count, ptr_device_vertices, obj.vertex_count, scene_bounding_box);

  printf("Constr (us)\tRender (us)\n"); //Print benchmark output table headers

  //Depending on user choice, construct BVH.
  Node* ptr_device_tree = nullptr;
  if(bvh_type == BVH_Type::LBVH)
    ptr_device_tree = lbvh.construct();   // Construct Karras 2012
  else if(bvh_type == BVH_Type::SAHBVH) 
    ptr_device_tree = sahbvh.construct(); // Construct Wald   2007


  // ----------- RENDER -----------
  if(render_type == Render_Type::NONE)
    return 0;

  RenderConfig config(image_width, image_height, samples_per_pixel, max_bounces, 1337);

  Camera cam = Camera(config.img_width, config.img_height, 90.0f, 1.0f, Vec3(cam_x, cam_y, cam_z));
  Raytracer raytracer = Raytracer(config, ptr_device_vertices, ptr_device_normals, ptr_device_triangles, obj.index_count);

  //Benchmark rendering
  std::chrono::steady_clock::time_point start = std::chrono::high_resolution_clock::now();

  Vec3* ptr_device_img;
  if(render_type == Render_Type::HEATMAP){
    Vec3* device_traversal_statistics = nullptr;
    ptr_device_img = raytracer.renderTraversalHeatmap(cam, ptr_device_tree, &device_traversal_statistics);
    
    int image_size = config.img_width * config.img_height;
    Vec3* host_traversal_statistics = (Vec3*)malloc(image_size * sizeof(Vec3));
    checkCudaErrors(hipMemcpy(host_traversal_statistics, device_traversal_statistics, image_size * sizeof(Vec3), hipMemcpyDeviceToHost));
    FILE *fp = fopen("traversal_steps.txt", "w");
    for (int i = 0; i < config.img_width * config.img_height; i++) {
      int steps = (int)host_traversal_statistics[i].x();
      fprintf(fp, "%i\n", steps);
    }
    fprintf(fp, "\n");
    fclose(fp);
  }
  else{
    if(bvh_type == BVH_Type::NONE)
      ptr_device_img = raytracer.render(cam);
    else
      ptr_device_img = raytracer.render(cam, ptr_device_tree);
  }
  
  std::chrono::steady_clock::time_point stop = std::chrono::high_resolution_clock::now();
  long long duration_us = std::chrono::duration_cast<std::chrono::duration<long long, std::micro>>(stop - start).count();
  printf("\t\t%lli\n", duration_us);

  //Copy framebuffer from device to host and save to disk.
  Image render_output = Image(config.img_width, config.img_height);
  render_output.copyFromDevice(ptr_device_img, config.img_height * config.img_width);

  render_output.save(output_filename);

  hipFree(ptr_device_triangles);
  hipFree(ptr_device_vertices);
  hipFree(ptr_device_normals);
  return 0;
}