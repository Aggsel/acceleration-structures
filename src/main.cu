#include "hip/hip_runtime.h"
#define TINYOBJLOADER_IMPLEMENTATION 
#define PI 3.1415926535897932385
#define EPSILON 0.000001

#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <limits.h>

#include "node.h"
#include "triangle.h"
#include "render_config.h"
#include "vec3.h"
#include "vec2.h"
#include "ray.h"
#include "hit.h"
#include "camera.h"
#include "main.h"
#include "math_util.h"

#include "cuda_helpers/hip/hip_runtime_api.h"      //checkCudaErrors
#include "tiny_obj_loader.h"

__device__ int2 determineRange(Triangle *sorted_morton_codes, int total_primitives, int node_index){

  if(node_index == 0){
    int2 range;
    range.x = 0;
    range.y = total_primitives-1;
    return range;
  }

  //Determine direction (d).
  int current_code = sorted_morton_codes[node_index].morton_code;

  //BUG: Properly handle out of bounds exceptions. (branchless? :O)

  int prev_code = sorted_morton_codes[node_index-1].morton_code;
  int next_code = sorted_morton_codes[node_index+1].morton_code;

  if(prev_code == current_code)
    prev_code = prev_code ^ node_index-1;
  if(next_code == current_code)
    next_code = next_code ^ node_index-1;

  int next_delta = __clz(current_code ^ next_code);
  int prev_delta = __clz(current_code ^ prev_code);
  int d = next_delta - prev_delta < 0 ? -1 : 1;

  //Compute upper bound for the length of the range.
  // int lmax = 2;
  // int delta_min = __clz(current_code ^ sorted_morton_codes[node_index-d].morton_code);
  // while(__clz(current_code ^ sorted_morton_codes[node_index + lmax * d].morton_code) > delta_min){
  //   lmax *= 2;
  // }

  int lmax = 2;
  int delta_min = next_delta < prev_delta ? next_delta : prev_delta;
  int delta = -1;
  int i = node_index + d * lmax;
  if(0 <= i && i < total_primitives){
      delta = __clz(node_index ^ sorted_morton_codes[i].morton_code);
  }
  while(delta > delta_min){
      lmax = lmax << 1;
      i = node_index + d * lmax;
      delta = -1;
      if(0 <= i && i < total_primitives){
          delta = __clz(node_index ^ sorted_morton_codes[i].morton_code);
      }
  }

  //Binary search, other end.
  // int l = 0;
  // int step = lmax;
  
  // do{
  //   step = (step + 1) >> 1; // exponential decrease
  //   int comparing_index = node_index + (l + step) * d;
  //   if(__clz(current_code ^ sorted_morton_codes[comparing_index].morton_code) > delta_min)
  //     l = l + step;
  // }while (step > 1);
  // int j = node_index + l * d;
  int l = 0;
  int t = lmax >> 1;
  while(t > 0){
      i = node_index + (l + t) * d;
      delta = -1;
      if(0 <= i && i < total_primitives){
          delta = __clz(current_code ^ sorted_morton_codes[i].morton_code);
      }
      if(delta > delta_min){
          l += t;
      }
      t >>= 1;
  }
  unsigned int j = node_index + l * d;

  int2 min_max;
  min_max.x = min(node_index, j);
  min_max.y = max(node_index, j);
  return min_max;
}

__device__ int findSplit(Triangle *sorted_morton_codes, int first, int last){
  int first_morton = sorted_morton_codes[first].morton_code;
  int last_morton = sorted_morton_codes[last].morton_code;

  if(first_morton == last_morton)
    return (first + last) >> 1;

  //count leading zeros
  int common_prefix = __clz(first_morton ^ last_morton);

  // Use binary search to find where the next bit differs.
  // Specifically, we are looking for the highest object that
  // shares more than common_prefix bits with the first one.
  int split = first;
  int step = last - first;

  do{
    step = (step + 1) >> 1;
    int new_split = split + step;

    if (new_split < last){
      int split_morton = sorted_morton_codes[new_split].morton_code;
      int split_prefix = __clz(first_morton ^ split_morton);
      if (split_prefix > common_prefix)
        split = new_split;
    }
  }while (step > 1);
  
  return split;
}

__global__ void constructHLBVH(Triangle *triangles, Node* internalNodes, int primitive_count){
  int node_index = blockIdx.x *blockDim.x + threadIdx.x;
  if(node_index >= primitive_count)
    return;

  // Find out which range of objects the node corresponds to.
  // (This is where the magic happens!)

  //binary search morton codes.
  int2 range = determineRange(triangles, primitive_count, node_index);
  int first = range.x;
  int last = range.y;

  // Determine where to split the range.
  int split = findSplit(triangles, first, last);
  //BUG: Split is sometimes a really large value.

  printf("Node: %i, \tMorton: %i, \tMin: %i, \tMax: %i, \tSplit: %i\n", node_index, triangles[node_index].morton_code, first, last, split);
  // printf("Node: %i, \tMin: %i, \tMax: %i, \tSplit: %i\n", node_index, first, last, split);

  if(split >= primitive_count)
    return;

  // Select left_child.
  Node* left_child = &internalNodes[split];
  if (split == first){
    left_child->primitive = &triangles[split];
    left_child->isLeaf = true;
  }
  else{
    left_child = &internalNodes[split];
  }

  // Select rightChild.
  Node* right_child = &internalNodes[split + 1];
  if (split + 1 == last){
    right_child->primitive = &triangles[split];
    right_child->isLeaf = true;
  }
  else{
    right_child = &internalNodes[split + 1];
  }

  // Record parent-child relationships.
  internalNodes[node_index].leftChild = left_child;
  internalNodes[node_index].rightChild = right_child;
  left_child->parent = &internalNodes[node_index];
  right_child->parent = &internalNodes[node_index];

  // Node 0 is always the root of the tree, the pointer supplied (Node* internalNodes) will be the address of the root.
}

__global__ void render(Vec3 *output_image, Camera cam, hiprandState *rand, RenderConfig config, Vec3 *vertices, Triangle *triangles, int vertex_count, Vec3 *normals){
  int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
  int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
  if((pixel_x >= config.img_width) || (pixel_y >= config.img_height)) return;
  int pixel_index = pixel_y*config.img_width + pixel_x;

  hiprandState local_rand = rand[pixel_index];

  Vec3 result = Vec3(0.0, 0.0, 0.0);
  for (int i = 0; i < config.samples_per_pixel; i++){
    Vec2 uv = Vec2((pixel_x + hiprand_uniform(&local_rand)) / (config.img_width-1), (pixel_y+ hiprand_uniform(&local_rand)) / (config.img_height-1));
    Ray ray = Ray(Vec3(0,0,0), normalize(cam.lower_left_corner + uv.x()*cam.horizontal + uv.y()*cam.vertical - Vec3(0,0,0)) );
    Vec3 out_col = color(&ray, &local_rand, config.max_bounces, vertices, triangles, vertex_count, normals);

    float r = clamp01(out_col.x());
    float g = clamp01(out_col.y());
    float b = clamp01(out_col.z());
    result = result + Vec3(r,g,b);
  }
  
  //Gamma correction
  float scale = 1.0 / config.samples_per_pixel;
  float r = sqrt(scale * result.x());
  float g = sqrt(scale * result.y());
  float b = sqrt(scale * result.z());
  result = Vec3(r,g,b);

  output_image[pixel_index] = result;
}

//As mentioned in Accelerated Ray Tracing in One Weekend (https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/)
//It's a good idea to seperate initialization and actual rendering if we want accurate performance numbers. 
__global__ void initKernels(int image_width, int image_height, unsigned long long rand_seed, hiprandState *rand){
  int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
  int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
  if((pixel_x >= image_width) || (pixel_y >= image_height)) return;
  int pixel_index = pixel_y*image_width + pixel_x;

  hiprand_init(rand_seed, pixel_index, 0, &rand[pixel_index]);
}

__device__ Vec3 color(Ray *ray, hiprandState *rand, int max_depth, Vec3 *vertices, Triangle *triangles, int vertex_count, Vec3 *normals) {
  float cur_attenuation = 1.0f;
  for(int i = 0; i < max_depth; i++) {
    RayHit hit;
    bool was_hit = false;
    for (int j = 0; j < vertex_count/3; j++){
      RayHit tempHit;

      if (!intersectTri(ray, &tempHit,  vertices[triangles[j].v0_index],
                                        vertices[triangles[j].v1_index],
                                        vertices[triangles[j].v2_index],
                                        normals[triangles[j].v0_index],
                                        normals[triangles[j].v1_index],
                                        normals[triangles[j].v2_index]))
        continue; //Did not hit triangle.
      
      if(tempHit.dist > hit.dist)
        continue; //Hit triangle but not closest intersection so far.

      hit.dist = tempHit.dist;
      hit.normal = tempHit.normal;
      hit.pos = tempHit.pos;
      hit.uv = tempHit.uv;
      was_hit = true;
    }

    if(was_hit){
      Vec3 target = hit.pos + hit.normal + randomInUnitSphere(rand);
      cur_attenuation *= 0.5f;
      ray->org = hit.pos;
      ray->dir = normalize(target - hit.pos);
      continue;
    }
    else {
      Vec3 unit_direction = normalize(ray->direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);
      Vec3 c = (1.0f-t)*Vec3(1.0, 1.0, 1.0) + t*Vec3(0.5, 0.7, 1.0);
      return cur_attenuation * c;
    }
  }
  return Vec3(0.0, 0.0, 0.0);
}

__device__ Vec3 randomInUnitSphere(hiprandState *rand){
  while(true){
    float x = (hiprand_uniform(rand) * 2.0) - 1.0;
    float y = (hiprand_uniform(rand) * 2.0) - 1.0;
    float z = (hiprand_uniform(rand) * 2.0) - 1.0;
    Vec3 p = Vec3(x, y, z);
    if(sqrMagnitude(p) >= 1)
      continue;
    return p;
  }
}

/* From Möller & Trumbore, Fast, Minimum Storage Ray/Triangle Intersection */
__device__ bool intersectTri(Ray *ray, RayHit *bestHit, Vec3 v0, Vec3 v1, Vec3 v2, Vec3 n0, Vec3 n1, Vec3 n2){
  Vec3 edge1 = v1 - v0;
  Vec3 edge2 = v2 - v0;

  Vec3 pvec = cross(ray->direction(), edge2);
  float det = dot(edge1, pvec);
  //Culling implementation
  if(det < EPSILON)
    return false;
  
  Vec3 tvec = ray->origin() - v0;
  bestHit->uv.e[0] = dot(tvec, pvec);
  if(bestHit->uv.x() < 0.0 || bestHit->uv.x() > det)
    return false;

  Vec3 qvec = cross(tvec, edge1);
  bestHit->uv.e[1] = dot(ray->direction(), qvec);
  if(bestHit->uv.y() < 0.0 || bestHit->uv.x() + bestHit->uv.y() > det)
    return false;

  float inv_det = 1.0 / det;
  bestHit->dist = dot(edge2, qvec) * inv_det;
  bestHit->uv.e[0] *= inv_det;
  bestHit->uv.e[1] *= inv_det;
  bestHit->normal = normalize(cross(edge1, edge2));
  bestHit->pos = ray->point_along_ray(bestHit->dist);

  //BUG: There's something funky with the normals when interpolating...
  // bestHit->normal = normalize(bestHit->uv.x()*n1 + bestHit->uv.y() * n2 + (1.0 - bestHit->uv.x() - bestHit->uv.y()) * n0);
  return true;
}

//From Shirleys Ray Tracing in One Weekend.
__device__ bool intersectSphere(Ray *ray, RayHit *bestHit, Vec3 point, float radius){
  Vec3 oc = ray->origin() - point;
  float a = sqrMagnitude(ray->direction());
  float half_b = dot(oc, ray->direction());
  float c = sqrMagnitude(oc) - radius*radius;

  float discriminant = half_b*half_b - a*c;
  if (discriminant < 0) return false;
  float sqrtd = sqrt(discriminant);

  float root = (-half_b - sqrtd) / a;
  if (root < 0.00001 || 999999.0 < root) {
      root = (-half_b + sqrtd) / a;
      if (root < 0.00001 || 999999.0 < root)
          return false;
  }
  bestHit->dist = root;
  bestHit->pos = ray->point_along_ray(bestHit->dist);
  bestHit->normal = (bestHit->pos - point) / radius;
  return true;
}

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ __host__ inline unsigned int expandBits(unsigned int v){
  v = (v * 0x00010001u) & 0xFF0000FFu;
  v = (v * 0x00000101u) & 0x0F00F00Fu;
  v = (v * 0x00000011u) & 0xC30C30C3u;
  v = (v * 0x00000005u) & 0x49249249u;
  return v;
}

//Expects an input Vec3(0..1, 0..1, 0..1)
__device__ __host__ int mortonCode(Vec3 v){
  //Clamp coordinates to 10 bits.
  float x = min(max(v.x() * 1024.0f, 0.0f), 1023.0f);
  float y = min(max(v.y() * 1024.0f, 0.0f), 1023.0f);
  float z = min(max(v.z() * 1024.0f, 0.0f), 1023.0f);
  //Bit shift componentwise before merging bits into morton code.
  unsigned int xx = expandBits((unsigned int)x) << 2;
  unsigned int yy = expandBits((unsigned int)y) << 1;
  unsigned int zz = expandBits((unsigned int)z);
  return xx | yy | zz;
}

int serializeImageBuffer(Vec3 *ptr_img, const char *file_name, int image_width, int image_height){
  FILE *fp = fopen(file_name, "w");
  fprintf(fp, "P3\n%d %d\n255\n", image_width, image_height);

  for (int j = image_height-1; j >= 0; j--) {
    for (int i = 0; i < image_width; i++) {
      size_t pixel_index = j*image_width + i;
      float r = clamp01(abs(ptr_img[pixel_index].x()));
      float g = clamp01(abs(ptr_img[pixel_index].y()));
      float b = clamp01(abs(ptr_img[pixel_index].z()));
      int ir = int(255.99*r);
      int ig = int(255.99*g);
      int ib = int(255.99*b);
      fprintf(fp, "%d %d %d\n", ir, ig, ib);
    }
  }

  fclose(fp);
  return 0;
}

int main(int argc, char *argv[]){
  std::string filename = "sample_models/test2.obj";

  tinyobj::ObjReaderConfig reader_config;
  tinyobj::ObjReader reader;

  if (!reader.ParseFromFile(filename, reader_config)) {
    if (!reader.Error().empty()) {
        std::cerr << "TinyObjReader: " << reader.Error();
    }
    exit(1);
  }

  if (!reader.Warning().empty()) {
    std::cout << "TinyObjReader: " << reader.Warning();
  }

  const tinyobj::attrib_t &attrib = reader.GetAttrib();
  const std::vector<tinyobj::shape_t> &shapes = reader.GetShapes();

  std::cout << "\nFile '" << filename << "' loaded." << std::endl;
  int vertex_count = (int)(attrib.vertices.size()) / 3;
  printf("# vertices        = %d\n", vertex_count);
  int indices_count = (int)(shapes[0].mesh.indices.size());
  printf("# vertex indices  = %d\n", indices_count);
  int normals_count = (int)(attrib.normals.size()) / 3;
  printf("# normals         = %d\n\n", normals_count);


  //Calculate scene bounding box.
  Vec3 min_bounds = Vec3( 10000000.0, 10000000.0,   10000000.0);
  Vec3 max_bounds = Vec3(-10000000.0,-10000000.0,  -10000000.0);

  for (int i = 0; i < vertex_count; i+=3){
    min_bounds.e[0] = min(min_bounds.x(), attrib.vertices[i  ]);
    min_bounds.e[1] = min(min_bounds.y(), attrib.vertices[i+1]);
    min_bounds.e[2] = min(min_bounds.z(), attrib.vertices[i+2]);

    max_bounds.e[0] = max(max_bounds.x(), attrib.vertices[i  ]);
    max_bounds.e[1] = max(max_bounds.y(), attrib.vertices[i+1]);
    max_bounds.e[2] = max(max_bounds.z(), attrib.vertices[i+2]);
  }
  //BUG: Calculate these bounds, for some reason this does not work.
  min_bounds = Vec3(-245.425491, -99.999916, -1256.244751); //Only relevant for test2.obj
  max_bounds = Vec3(302.590363, 546.458801, -250.774368);
  printf("Min Bounds: (%f, %f, %f)\n", min_bounds.x(), min_bounds.y(), min_bounds.z());
  printf("Max Bounds: (%f, %f, %f)\n", max_bounds.x(), max_bounds.y(), max_bounds.z());
  Vec3 bounds = abs(min_bounds) + abs(max_bounds);
  printf("Bounds: (%f, %f, %f)\n\n", bounds.x(), bounds.y(), bounds.z());

  //The Obj reader does not store vertex indices in contiguous memory.
  //Copy the indices into a block of memory on the host device.
  Triangle *ptr_host_triangles = (Triangle*)malloc(sizeof(Triangle) * indices_count/3);
  for (int i = 0; i < indices_count; i+=3){
    Triangle tempTri = Triangle();
    int v0_index = shapes[0].mesh.indices[i  ].vertex_index;
    int v1_index = shapes[0].mesh.indices[i+1].vertex_index;
    int v2_index = shapes[0].mesh.indices[i+2].vertex_index;
    tempTri.v0_index = v0_index;
    tempTri.v1_index = v1_index;
    tempTri.v2_index = v2_index;

    //TODO @Perf: The morton code generation could easily be done on the GPU instead.
    tinyobj::index_t idx = shapes[0].mesh.indices[i];
    Vec3 v0 = Vec3( attrib.vertices[3*size_t(idx.vertex_index)+0], 
                    attrib.vertices[3*size_t(idx.vertex_index)+1], 
                    attrib.vertices[3*size_t(idx.vertex_index)+2] );

    idx = shapes[0].mesh.indices[i+1];
    Vec3 v1 = Vec3( attrib.vertices[3*size_t(idx.vertex_index)+0], 
                    attrib.vertices[3*size_t(idx.vertex_index)+1], 
                    attrib.vertices[3*size_t(idx.vertex_index)+2] );

    idx = shapes[0].mesh.indices[i+2];
    Vec3 v2 = Vec3( attrib.vertices[3*size_t(idx.vertex_index)+0], 
                    attrib.vertices[3*size_t(idx.vertex_index)+1], 
                    attrib.vertices[3*size_t(idx.vertex_index)+2] );

    Vec3 centroid = (v0 + v1 + v2) / 3;

    printf("Centroid: (%f, %f, %f)\n", centroid.x(), centroid.y(), centroid.z());
    centroid.e[0] = (centroid.x() - min_bounds.x()) / (max_bounds.x() - min_bounds.x());
    centroid.e[1] = (centroid.y() - min_bounds.y()) / (max_bounds.y() - min_bounds.y());
    centroid.e[2] = (centroid.z() - min_bounds.z()) / (max_bounds.z() - min_bounds.z());
    printf("Centroid: (%f, %f, %f)\n\n", centroid.x(), centroid.y(), centroid.z());
    tempTri.morton_code = mortonCode(centroid);
    ptr_host_triangles[i/3] = tempTri;
  }

  //Allocate and memcpy index, vertex and normal buffers from host to device.
  Triangle *ptr_device_triangles;
  checkCudaErrors(hipMalloc((void**)&ptr_device_triangles, indices_count/3 * sizeof(Triangle)));
  checkCudaErrors(hipMemcpy(ptr_device_triangles, ptr_host_triangles, indices_count/3 * sizeof(Triangle), hipMemcpyHostToDevice));
  
  Vec3 *ptr_device_vertices;
  checkCudaErrors(hipMalloc(&ptr_device_vertices, vertex_count * sizeof(Vec3)));
  checkCudaErrors(hipMemcpy(ptr_device_vertices, attrib.vertices.data(), vertex_count * sizeof(Vec3), hipMemcpyHostToDevice));

  Vec3 *ptr_device_normals;
  checkCudaErrors(hipMalloc(&ptr_device_normals, normals_count * sizeof(Vec3)));
  checkCudaErrors(hipMemcpy(ptr_device_normals, attrib.normals.data(), normals_count * sizeof(Vec3), hipMemcpyHostToDevice));

  //Set default values for filename and image size.
  char* output_filename = "output.ppm";
                    //w    h    spp   max_depth
  RenderConfig config(512, 512, 30, 5         );
  Camera cam = Camera(config.img_width, config.img_height, 90.0f, 1.0f, Vec3(0,0,0));

  hiprandState *d_rand_state;
  Vec3 *ptr_img;
  checkCudaErrors(hipMalloc(&d_rand_state, config.img_width * config.img_height*sizeof(hiprandState)));
  checkCudaErrors(hipMallocManaged(&ptr_img, config.img_width * config.img_height*sizeof(Vec3)));  //BUG: Segmentation Fault when using unmanaged malloc.

  // ---------- SORT -----------
  // Sorts the triangle buffer based on the computed morton codes. (using < overloading from the triangle struct).
  thrust::sort(thrust::device, ptr_device_triangles, ptr_device_triangles+indices_count/3);    
  checkCudaErrors(hipMemcpy(ptr_host_triangles, ptr_device_triangles, indices_count/3 * sizeof(Triangle), hipMemcpyDeviceToHost));

  // ---------- CONSTRUCT -----------
  int primitive_count = indices_count/3;
  Node* ptr_device_internal_nodes;
  checkCudaErrors(hipMalloc(&ptr_device_internal_nodes, (primitive_count-1)*sizeof(Node)));
  printf("Primitives: %i, Thread blocks: %i, Threads per block: %i \n", primitive_count, primitive_count/64+1, 64);
  // <<<x,y>>> Launches x thread blocks with y threads per block.
  constructHLBVH<<<primitive_count/64+1,64>>>(ptr_device_triangles, ptr_device_internal_nodes, primitive_count);
  checkCudaErrors(hipDeviceSynchronize());

  // ---------- RENDER -------------
  int threads_x = 8;
  int threads_y = 8;
  dim3 threads(threads_x,threads_y);
  dim3 tracingBlocks(config.img_width/threads_x+1,config.img_height/threads_y+1);

  printf("Initializing kernels... ");
  initKernels<<<tracingBlocks, threads>>>(config.img_width, config.img_height, 1337, d_rand_state);
  checkCudaErrors(hipDeviceSynchronize());
  
  printf("Initialization complete.\nStarting Rendering... ");
  render<<<tracingBlocks, threads>>>(ptr_img, cam, d_rand_state, config, ptr_device_vertices, ptr_device_triangles, indices_count, ptr_device_normals);
  checkCudaErrors(hipDeviceSynchronize());

  printf("Render complete.\nWriting to disk... ");
  serializeImageBuffer(ptr_img, output_filename, config.img_width, config.img_height);
  printf("Saved to disk.\n");

  free(ptr_host_triangles);
  checkCudaErrors(hipFree(ptr_device_internal_nodes));
  checkCudaErrors(hipFree(ptr_img));
  checkCudaErrors(hipFree(d_rand_state));
  checkCudaErrors(hipFree(ptr_device_triangles));
  checkCudaErrors(hipFree(ptr_device_vertices));
  checkCudaErrors(hipFree(ptr_device_normals));
  return 0;
}